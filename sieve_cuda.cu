/*
-----------------------------------------------------------------------------------
File name     : main.cpp
Lab name      : "06 Crible" or "Sieve"
Author(s)     : Leandro Saraiva Maia, Miguel Jalube
Creation date : 18.11.2021

Description   : This programs asks the user for an amount of prime numbers to
                find. Then, it uses the sieve of Eratosthenes to compute the
                amount of prime numbers wanted (start from 0 -> amount). Finally,
                it displays sieve table and the prime that have been found.
Remark(s)     : The clion console is a bit bugged and may display values
                incorrectly, it is better to use an external console.
                This program is meant to be used only in english, there are no
                constants sets to facilitate potential translation.

 
Compiler      : Mingw-w64 g++ 11.1.0
-----------------------------------------------------------------------------------
*/

#include <cstdlib>   
#include <iostream>  
#include <limits>
#include <string>    
#include <iomanip>   
#include <cassert>   
#include <cmath>     
#include <time.h>
#include <hip/hip_runtime.h>

// taille du tableau = quantité de nombre que le bloc peut vérifier si il est prmier ou non 
// Ne pas dépasser 40000 car il y a une limite à la mémoire partagée par bloc
#define SHARE_MEMORY_SIZE 40000

using namespace std;

unsigned getNumberOfDigits(const unsigned long long number);
void setAllElementsInArray(bool arrayToSet[], const size_t arrSize,
                           const bool defaultValue);
void setArrayWithAscendingOrder(unsigned long long arrayToSet[], const size_t arrSize,
                                const size_t startsAt);
void displayArrayAsTable(const unsigned long long array[], const size_t arrSize,
                         const unsigned nbCol, const int colWidth);
void sieve(bool sieveArray[], size_t sieveArraySize);
unsigned long long extractPrimeNumbers(const bool sieveArray[], size_t sieveArraySize,
								     unsigned long long primeArray[]);

int main(int argc, char* argv[]) {
   clock_t start_time = clock();

	//---------- Variables and constants ----------
	const unsigned long long   MIN_PRIME_UP_TO =                2ull;
   const unsigned long long   MAX_PRIME_UP_TO =                100000ull; // >100000 may cause memory issues
   const unsigned             RESULT_NB_COL =                  20u;
	unsigned long long         primeNumArray[MAX_PRIME_UP_TO];
   unsigned long long         numToCheckForPrime;
	unsigned long long         numOfPrimeNumbers;
	bool                       sieveArray[MAX_PRIME_UP_TO];


   if(argc != 2 || stoull(argv[1]) < MIN_PRIME_UP_TO || stoull(argv[1]) > MAX_PRIME_UP_TO){
      cout << "Usage : ./sieve <number>" << endl;
      cout << MIN_PRIME_UP_TO << " < [number] < " << MAX_PRIME_UP_TO << endl;
      return EXIT_FAILURE;
   }

   numToCheckForPrime = stoull(argv[1]);

   //---------- Initialization ----------
   setArrayWithAscendingOrder(primeNumArray, numToCheckForPrime, 1);
   setAllElementsInArray(sieveArray, (size_t)numToCheckForPrime, true);

   clock_t init_time = clock();

   //---------- Computes prime numbers with Eratosthenes sieve ----------
   sieve(sieveArray, numToCheckForPrime);

   clock_t sieve_time = clock();

   numOfPrimeNumbers = extractPrimeNumbers(sieveArray, numToCheckForPrime, primeNumArray);

   clock_t extract_time = clock();

   //---------- Display the sieve results ----------
   cout << endl << numOfPrimeNumbers << " prime number(s) found up to " << numToCheckForPrime << endl;
   displayArrayAsTable(primeNumArray, numOfPrimeNumbers, RESULT_NB_COL, int(getNumberOfDigits(MAX_PRIME_UP_TO)));
   cout << endl;

   clock_t display_time = clock();

   //---------- Display time measures --------------
   printf("============= Time measures ===============\n");
   printf("Init time :     %f\n", (double)(init_time - start_time) / CLOCKS_PER_SEC);
   printf("Sieve time :    %f\n", (double)(sieve_time - init_time) / CLOCKS_PER_SEC);
   printf("Extract time :  %f\n", (double)(extract_time - sieve_time) / CLOCKS_PER_SEC);
   printf("Display time :  %f\n", (double)(display_time - extract_time) / CLOCKS_PER_SEC);
   printf("Total time :    %f\n", (double)(display_time - start_time) / CLOCKS_PER_SEC);

   return EXIT_SUCCESS;
}

unsigned getNumberOfDigits(const unsigned long long number){
   return number > 0ull ? unsigned(log10(double(number))) + 1ull : 1ull;
}

// We could imagine an overload for other data types
void setAllElementsInArray(bool arrayToSet[], const size_t arrSize,
                           const bool defaultValue) {
   // Check if array is not null
   assert(arrayToSet != nullptr);
   for (size_t i = 0ull; i < arrSize; ++i) {
      arrayToSet[i] = defaultValue;
   }
}

void setArrayWithAscendingOrder(unsigned long long arrayToSet[], const size_t arrSize,
                                const size_t startsAt = 0ull) {
   // Check if array is not null
   assert(arrayToSet != nullptr);
   for (size_t i = 0ull; i < arrSize; ++i) {
      arrayToSet[i] = i + startsAt;
   }
}

void displayArrayAsTable(const unsigned long long array[], const size_t arrSize,
                         const unsigned nbCol, const int colWidth) {
   assert(array != nullptr);
   for (size_t i = 0ull; i < arrSize; ++i) {
      cout << setw(colWidth) << array[i];
      if ((i + 1ull) % (unsigned long long)nbCol == 0ull && i != arrSize - 1ull)
         cout << endl;
   }
}

void displayArrayAsTable(const bool array[], const unsigned long arrSize,
                         const unsigned nbCol, const int colWidth,
                         const char valueWhenFalse, const char valueWhenTrue) {
   assert(array != nullptr);
   for (size_t i = 0ull; i < arrSize; ++i) {
      cout << setw(colWidth) << (array[i] ? valueWhenTrue : valueWhenFalse);
      if ((i + 1ull) % (unsigned long long)nbCol == 0ull && i != arrSize - 1ull)
         cout << endl;
   }
}

__global__ void kernel(size_t sieveArraySize, bool *sieveArray) {

   unsigned int thread_id = threadIdx.x;

   __shared__ bool sharedSieveArray[SHARE_MEMORY_SIZE];

   // défini le nombre d'itération du crible d'hératostène
   int threadRangeNb = (sieveArraySize + blockDim.x - 1) / blockDim.x;

   // défini la quantité de nombre que le bloc va vérifier (doit être )
   int blockRangeNb = (sieveArraySize + gridDim.x - 1) / gridDim.x;
   assert(SHARE_MEMORY_SIZE >= blockRangeNb);

   // Tous les nombres sont initialisé comme premiers 
   for (int i = 0; i < blockRangeNb; ++i) {
      sharedSieveArray[i] = true;
   }

   __syncthreads();
   
   // Pour toute les itérations du crible de ce treads
   for (int i = 0; i < threadRangeNb; ++i) {

      //nombre de l'itéreation du crible
      int n = threadRangeNb * thread_id + i + 1;

      if(n == 1){
         // il ne faut pas tenter de diviser les nombres par 1
         continue;
      }else if (n < sieveArraySize) { // pourrait être opimisé en remplacant par racine de n mais nous gardons la structure originale du code

         // itére uniquement sur les nombres du bloc
         for (size_t j = max(n, blockIdx.x * blockRangeNb); j < min((size_t)(blockRangeNb * (blockIdx.x+1)), sieveArraySize); ++j) {
            // retires les nombres qui ne sont pas premiers
            if((j + 1ull) % n == 0ull){
               sharedSieveArray[j-blockIdx.x * blockRangeNb] = false;
            }
         }
      }
   }
   __syncthreads();

   // copie les nombres depuis la mémoire partagée
   for (int i = blockIdx.x * blockRangeNb; i < (blockIdx.x+1) * blockRangeNb; ++i) {
      sieveArray[i] = sharedSieveArray[i-blockIdx.x * blockRangeNb];
   }
}





void sieve(bool sieveArray[], size_t sieveArraySize){

   // 1 n'est pas premier
   sieveArray[0] = false;

   assert(sieveArray != nullptr && sieveArray != nullptr);

   size_t size = sieveArraySize * sizeof(bool);

   bool *d_sieveArray;
   hipMalloc((void **)&d_sieveArray, size);

   int blockSize = 256;
   int numBlocks = 4; // attention (sieveArray / numBlocks) doiit être supérieur à 40000 sinon problème de mémoire sur la gpu

   kernel<<<numBlocks, blockSize>>>(sieveArraySize, d_sieveArray);

   hipMemcpy(sieveArray, d_sieveArray, size, hipMemcpyDeviceToHost);
   
   hipFree(d_sieveArray);
}

unsigned long long extractPrimeNumbers(const bool sieveArray[], size_t sieveArraySize,
								     unsigned long long primeArray[]) {
	unsigned long long numOfPrimeNumbers = 0ull;
	for (size_t i = 0ull; i < sieveArraySize; ++i) {
		if (sieveArray[i]) {
			primeArray[numOfPrimeNumbers++] = i + 1ull;
		}
	}

	return numOfPrimeNumbers;
}